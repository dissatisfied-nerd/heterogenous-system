#include "balancing.cuh"

void demonstratePipelining() {
    const int numStages = 3;
    const int numElements = 10000;
    size_t size = numElements * sizeof(float);
    
    std::vector<std::thread> pipelineThreads;
    std::vector<float*> stageBuffers(numStages + 1);
    
    for (int i = 0; i <= numStages; ++i) {
        hipHostMalloc(&stageBuffers[i], size);
    }
    
    initializeVector(stageBuffers[0], numElements);
    
    for (int stage = 0; stage < numStages; ++stage) {
        pipelineThreads.emplace_back([stage, numElements, size, &stageBuffers]() {
            float *d_input, *d_output;
            hipMalloc(&d_input, size);
            hipMalloc(&d_output, size);
            
            hipMemcpyAsync(d_input, stageBuffers[stage], size, hipMemcpyHostToDevice);
            
            int threadsPerBlock = 256;
            int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
            vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_input, d_output, numElements);
            
            hipMemcpyAsync(stageBuffers[stage+1], d_output, size, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            
            hipFree(d_input);
            hipFree(d_output);
        });
    }
    
    for (auto& t : pipelineThreads) {
        t.join();
    }
    
    for (auto& buf : stageBuffers) {
        hipHostFree(buf);
    }
    
    std::cout << "Pipelining completed with " << numStages << " stages." << std::endl;
}