#include "memory.cuh"

std::vector<double> matrixMulPinned(const std::vector<double>& A, const std::vector<double>& B, int M, int K, int N) {
    if(A.size() != M*K || B.size() != K*N)
        throw std::invalid_argument("Invalid matrix dimensions");

    double *h_A, *h_B, *h_C;
    CUDA_CHECK(hipHostAlloc(&h_A, M*K*sizeof(double), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_B, K*N*sizeof(double), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_C, M*N*sizeof(double), hipHostMallocDefault));

    std::copy(A.begin(), A.end(), h_A);
    std::copy(B.begin(), B.end(), h_B);

    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, M*K*sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, K*N*sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, M*N*sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, M*K*sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K*N*sizeof(double), hipMemcpyHostToDevice));

    dim3 blocks((N + BLOCK_SIZE-1)/BLOCK_SIZE, (M + BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    matrixMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, M, N, K);

    CUDA_CHECK(hipMemcpy(h_C, d_C, M*N*sizeof(double), hipMemcpyDeviceToHost));
    
    std::vector<double> result(h_C, h_C + M*N);
    
    hipHostFree(h_A); 
    hipHostFree(h_B); 
    hipHostFree(h_C);
    hipFree(d_A); 
    hipFree(d_B); 
    hipFree(d_C);
    
    return result;
}