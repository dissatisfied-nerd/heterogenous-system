#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

const int N = 512;
#define BLOCK_SIZE 16

void profile(const char* label, void (*func)()) 
{
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    double time = std::chrono::duration<double, std::milli>(end - start).count();
    printf("[%s] Time: %.3f ms\n", label, time);
}

__global__ void matMulKernel(float* A, float* B, float* C, int N) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) 
    {
        float val = 0;
        
        for (int k = 0; k < N; ++k){
            val += A[row * N + k] * B[k * N + col];
        }
        
        C[row * N + col] = val;
    }
}

void run_explicit_memory() 
{
    size_t size = N * N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for (int i = 0; i < N * N; ++i) { h_A[i] = 1.0f; h_B[i] = 2.0f; }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
}

void run_unified_memory() 
{
    size_t size = N * N * sizeof(float);
    float *A, *B, *C;

    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    for (int i = 0; i < N * N; ++i) 
    { 
        A[i] = 1.0f; 
        B[i] = 2.0f; 
    }

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

    matMulKernel<<<blocks, threads>>>(A, B, C, N);

    hipDeviceSynchronize();
    hipFree(A); hipFree(B); hipFree(C);
}

void run_zero_copy() 
{
    size_t size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    
    hipSetDeviceFlags(hipDeviceMapHost);

    hipHostAlloc(&h_A, size, hipHostMallocMapped);
    hipHostAlloc(&h_B, size, hipHostMallocMapped);
    hipHostAlloc(&h_C, size, hipHostMallocMapped);

    hipHostGetDevicePointer(&d_A, h_A, 0);
    hipHostGetDevicePointer(&d_B, h_B, 0);
    hipHostGetDevicePointer(&d_C, h_C, 0);
    
    for (int i = 0; i < N * N; ++i) 
    { 
        h_A[i] = 1.0f; 
        h_B[i] = 2.0f; 
    }
    
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);
    
    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
}

void run_pinned_memory() 
{
    size_t size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    hipHostAlloc(&h_A, size, hipHostMallocDefault);
    hipHostAlloc(&h_B, size, hipHostMallocDefault);
    hipHostAlloc(&h_C, size, hipHostMallocDefault);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    for (int i = 0; i < N * N; ++i) 
    { 
        h_A[i] = 1.0f; 
        h_B[i] = 2.0f; 
    }
    
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice);
    
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);
    
    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
}

int main() {
    profile("Explicit Memory", run_explicit_memory);
    profile("Unified Memory", run_unified_memory);
    profile("Zero-Copy", run_zero_copy);
    profile("Pinned Memory", run_pinned_memory);
    return 0;
}
