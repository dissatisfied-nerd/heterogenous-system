#include "hip/hip_runtime.h"
#include "matrix.cuh"

__global__ void matrixMultiplyKernel(double* A, double* B, double* C, int rowsA, int colsA, int colsB)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0.0;
        for (int k = 0; k < colsA; ++k) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

void CUDAMultiply(const Matrix& A, const Matrix& B, Matrix& res)
{
    int rowsA = A.size();
    int colsA = A[0].size();
    int rowsB = B.size();
    int colsB = B[0].size();

    if (colsA != rowsB) {
        throw std::invalid_argument("Несовместимые размеры матриц для умножения.");
    }

    size_t sizeA = rowsA * colsA * sizeof(double);
    size_t sizeB = rowsB * colsB * sizeof(double);
    size_t sizeC = rowsA * colsB * sizeof(double);

    double *d_A, *d_B, *d_C;

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    std::vector<double> flatA(rowsA * colsA);
    std::vector<double> flatB(rowsB * colsB);
    for (int i = 0; i < rowsA; ++i)
        for (int j = 0; j < colsA; ++j)
            flatA[i * colsA + j] = A[i][j];
    for (int i = 0; i < rowsB; ++i)
        for (int j = 0; j < colsB; ++j)
            flatB[i * colsB + j] = B[i][j];

    hipMemcpy(d_A, flatA.data(), sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, flatB.data(), sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipDeviceSynchronize();

    std::vector<double> flatC(rowsA * colsB);
    hipMemcpy(flatC.data(), d_C, sizeC, hipMemcpyDeviceToHost);

    res.resize(rowsA, std::vector<double>(colsB));
    for (int i = 0; i < rowsA; ++i)
        for (int j = 0; j < colsB; ++j)
            res[i][j] = flatC[i * colsB + j];

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

float CUDAMultiplyProfile(const Matrix& A, const Matrix& B, Matrix& res)
{
    int rowsA = A.size();
    int colsA = A[0].size();
    int rowsB = B.size();
    int colsB = B[0].size();

    if (colsA != rowsB) {
        throw std::invalid_argument("Несовместимые размеры матриц для умножения.");
    }

    size_t sizeA = rowsA * colsA * sizeof(double);
    size_t sizeB = rowsB * colsB * sizeof(double);
    size_t sizeC = rowsA * colsB * sizeof(double);

    double *d_A, *d_B, *d_C;

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    std::vector<double> flatA(rowsA * colsA);
    std::vector<double> flatB(rowsB * colsB);
    
    for (int i = 0; i < rowsA; ++i){
        for (int j = 0; j < colsA; ++j){
            flatA[i * colsA + j] = A[i][j];
        }
    }

    for (int i = 0; i < rowsB; ++i){
        for (int j = 0; j < colsB; ++j){
            flatB[i * colsB + j] = B[i][j];
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(d_A, flatA.data(), sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, flatB.data(), sizeB, hipMemcpyHostToDevice);
    hipEventRecord(stop);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipDeviceSynchronize();

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    std::vector<double> flatC(rowsA * colsB);

    hipEventRecord(start1);
    hipMemcpy(flatC.data(), d_C, sizeC, hipMemcpyDeviceToHost);
    hipEventRecord(stop1);

    hipEventSynchronize(start1);
    hipEventSynchronize(stop1);
    float milliseconds1 = 0;
    hipEventElapsedTime(&milliseconds1, start1, stop1);

    res.resize(rowsA, std::vector<double>(colsB));
    
    for (int i = 0; i < rowsA; ++i){
        for (int j = 0; j < colsB; ++j){
            res[i][j] = flatC[i * colsB + j];
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    return milliseconds + milliseconds1;
}

